
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (100*1000*1000)

__global__ void vector_add(float *out, float *a, float *b, int n) {
  int index = threadIdx.x;
  int stride = blockDim.x;

  for(int i = index; i < n; i += stride) {
	 out[i] = a[i] + b[i];
  }
}

int main(){
	 float *a, *b, *out; // host
    float *d_a, *d_b, *d_out; // device

	 hipHostMalloc((void**)&a, sizeof(float) * N, hipHostMallocDefault);
    hipHostMalloc((void**)&b, sizeof(float) * N, hipHostMallocDefault);
    hipHostMalloc((void**)&out, sizeof(float) * N, hipHostMallocDefault);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

		hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	 for (int i = 0; i < 2; i++) {
		vector_add<<<1,256>>>(d_out, d_a, d_b, N);
	 }
		hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);


	 printf("vector[0] is %f\n", out[100]);

    // Cleanup after kernel execution
    hipFree(d_a);
	 hipFree(d_b);
	 hipFree(d_out);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(out);
}



/* __global__ void cuda_hello(){ */
/* } */

/* int main() { */
/*     cuda_hello<<<1,1>>>(); */

/*     return 0; */
/* } */
